
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define N (2048*2048)
#define THREAD_PER_BLOCK 512

__global__ void reverse(int* a, int* b){
 
 int index_in  = threadIdx.x + blockIdx.x * blockDim.x;	
 int index_out = gridDim.x * blockDim.x - index_in - 1;

 b[index_out] = a[index_in];
 
}

void random_ints(int *p, int n) {
	int i;
	for(i=0; i<n; i++) {
		p[i]=i;
	}
}


int main(void) {
 
 int *in, *out, *test;
 int *dev_in, *dev_out;
 int size = N*sizeof(int);
 int i;


 hipMalloc( (void**)&dev_in, size );
 hipMalloc( (void**)&dev_out, size );

 in  = (int*)malloc( size );
 out = (int*)malloc( size );
 test = (int*)malloc( size );

 random_ints(in, N );
 
 hipMemcpy( dev_in, in, size, hipMemcpyHostToDevice );
 
 reverse<<< N/THREAD_PER_BLOCK, THREAD_PER_BLOCK >>>(dev_in, dev_out);

 hipMemcpy(out, dev_out, size, hipMemcpyDeviceToHost);
 
 for(i = 0; i < N; i++){
	test[N-i-1] = in[i];
 }
 for(i = 0; i < N; i++){
	if(test[i]!= out[i]){
		printf("error: expected %d, got %d!\n",test[i], out[i]);
		break;
	}
 }
 
 if(i==N) {
   printf("correct!\n");
 }  


 free(in); free(out);

 hipFree(dev_in);
 hipFree(dev_out);


return 0;
}

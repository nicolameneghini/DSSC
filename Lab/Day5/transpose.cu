#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define N 10
#define N_THREADS 1
#define MAX_ELEM_VALUE 10
#define CACHE_BLOCK 32
#define ROWS 8

__global__ void transpose(float* mat, float *transp){


 //int y = blockIdx.x;
 //int x = threadIdx.x;

 //while(x < N){
  //transp[y*N + x] = mat[x*N + y];
  //x += blockDim.x;
 //}


  //transp[y*N + x] = mat[x*N + y];
  //x += blockDim.x;
 //}
 int index = threadIdx.x + blockIdx.x*blockDim.x;
 int x = index%N;
 int y = index/N;
 transp[y*N+x] = mat[x*N+y];
}

__global__ void shared_transpose(float* mat, float *transp){

 __shared__ float in_cache[CACHE_BLOCK*CACHE_BLOCK + 1];

 int x = blockIdx.x * CACHE_BLOCK + threadIdx.x;
 int y = blockIdx.y * CACHE_BLOCK + threadIdx.y;
 int width = gridDim.x * CACHE_BLOCK;

 for(unsigned int i = 0; i < CACHE_BLOCK; i+= ROWS)
        in_cache[(threadIdx.y+i)*CACHE_BLOCK + threadIdx.x] = mat[x + (y+i)*width];

 __syncthreads();

 x = blockIdx.x * CACHE_BLOCK + threadIdx.x;
 y = blockIdx.y * CACHE_BLOCK + threadIdx.y;

 for(unsigned int j = 0; j < CACHE_BLOCK; j+= ROWS)
         transp[x + (y+j)*width] = in_cache[threadIdx.x + CACHE_BLOCK*(threadIdx.y+j)];
}

v


void print_mat(int* mat){

 int i, j;

 for(i = 0; i < N; i++){
    for(j = 0; j < N; j++)
     printf("%d ",mat[i*N + j]);

    printf("\n");
  }
}

void randomly_fill_matrix(float *A)
{
  for (unsigned int i = 0; i < N; i++)
  {
    for (unsigned int j = 0; j < N; j++)
    {
      A[i*N + j] = rand() % (2 * MAX_ELEM_VALUE) - MAX_ELEM_VALUE;
    }
  }
}

void normal_transpose(float *mat, float *transp)
{
  for (unsigned int i = 0; i < N; i++)
          for (unsigned int j = 0; j < N; j++)
                  transp[i +j*N] = mat[i*N + j];
}


int test(float *a, float *b)
{

   for (unsigned int i = 0; i < N; i++){
      for (unsigned int j = 0; j < N; j++){
              if(b[i*N + j] != a[i*N + j])
                      return 0;
        }
   }

   return 1;
}

int main(void){

 float *mat, *transp1, *transp2, *test_mat;
 float *dev_mat, *dev_transp1, *dev_transp2;
 int size = N*N*sizeof(double);
 hipEvent_t start, stop;
 dim3 grid, block;
 block.x = CACHE_BLOCK;
 block.y = ROWS;
 grid.x = N/CACHE_BLOCK;
 grid.y = N/CACHE_BLOCK;

 hipMalloc( (void**)&dev_mat, size );
 hipMalloc( (void**)&dev_transp1, size );
 hipMalloc( (void**)&dev_transp2, size );
 mat = (float*)malloc(size);
 transp1 = (float*)malloc(size);
 transp2 = (float*)malloc(size);
 test_mat = (float*)malloc(size);



 randomly_fill_matrix(mat);
 print_mat(mat);

 normal_transpose(mat, test_mat);


 //----------//
 hipEventCreate(&start);
 hipEventCreate(&stop);

 hipMemcpy(dev_mat, mat, size, hipMemcpyHostToDevice );

 //-------------//
 hipEventRecord(start);
 transpose<<< (N*N)/N_THREADS, N_THREADS >>>(dev_mat, dev_transp1);
 hipEventRecord(stop);



 hipEventSynchronize(stop);
 float time_naive = 0;
 hipEventElapsedTime(&time_naive, start, stop);

 hipMemcpy(transp1, dev_transp1, size, hipMemcpyDeviceToHost);

 if(test(transp1, test_mat)) printf("correct1\n");
 else printf("not correct1\n");

 free(transp1); hipFree(dev_transp1);

 //-------------//

 hipEventRecord(start);
 shared_transpose<<< grid,block >>>(dev_mat, dev_transp2);
 hipEventRecord(stop);


 hipEventSynchronize(stop);
 float time_shared = 0;
 hipEventElapsedTime(&time_shared, start, stop);

 hipMemcpy(transp2, dev_transp2, size, hipMemcpyDeviceToHost);

 if(test(transp2, test_mat)) printf("correct2\n");
 else printf("not correct2\n");

 free(transp2); hipFree(dev_transp2);
 //-------------//


 print_mat(transp2);
 int n_threads = N_THREADS;
 printf("with %d threads time for a naive transpose is %fms while in shared memory is %fms\n", n_threads, time_naive, time_shared);

 free(mat); hipFree(dev_mat);

return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define N 8192
#define N_THREADS 1024
#define MAX_ELEM_VALUE 10
#define BLOCK 32
#define ROWS 8

__global__ void transpose(float* mat, float *transp){


 //int y = blockIdx.x;
 //int x = threadIdx.x;

 //while(x < N){
  //transp[y*N + x] = mat[x*N + y];
  //x += blockDim.x;
 //}


  //transp[y*N + x] = mat[x*N + y];
  //x += blockDim.x;
 //}
 int index = threadIdx.x + blockIdx.x*blockDim.x;
 int x = index%N;
 int y = index/N;
 transp[y*N+x] = mat[x*N+y];
}

__global__ void shared_transpose(float* mat, float *transp){

    __shared__ double in_cache[BLOCK][BLOCK+1];
    
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    
    in_cache[threadIdx.x][threadIdx.y] = mat[index_y * N + index_x];
    
    __syncthreads();
    
    transp[index_x * N + index_y] = in_cache[threadIdx.x][threadIdx.y];

}


void print_mat(float* mat){

 int i, j;

 for(i = 0; i < N; i++){
    for(j = 0; j < N; j++)
     printf("%f ",mat[i*N + j]);

    printf("\n");
  }
}

void randomly_fill_matrix(float *A)
{
  for (unsigned int i = 0; i < N; i++)
  {
    for (unsigned int j = 0; j < N; j++)
    {
      A[i*N + j] = rand() % (2 * MAX_ELEM_VALUE) - MAX_ELEM_VALUE;
    }
  }
}

void normal_transpose(float *mat, float *transp)
{
  for (unsigned int i = 0; i < N; i++)
          for (unsigned int j = 0; j < N; j++)
                  transp[i +j*N] = mat[i*N + j];
}


int test(float *a, float *b)
{

   for (unsigned int i = 0; i < N; i++){
      for (unsigned int j = 0; j < N; j++){
              if(b[i*N + j] != a[i*N + j])
                      return 0;
        }
   }

   return 1;
}

int main(void){

 float *mat, *transp1, *transp2, *test_mat;
 float *dev_mat, *dev_transp1, *dev_transp2;
 int size = N*N*sizeof(double);
 hipEvent_t start, stop;
 dim3 grid, block;
 block.x = BLOCK;
 block.y = BLOCK;
 grid.x = N/BLOCK;
 grid.y = N/BLOCK;

 hipMalloc( (void**)&dev_mat, size );
 hipMalloc( (void**)&dev_transp1, size );
 hipMalloc( (void**)&dev_transp2, size );
 mat = (float*)malloc(size);
 transp1 = (float*)malloc(size);
 transp2 = (float*)malloc(size);
 test_mat = (float*)malloc(size);



 randomly_fill_matrix(mat);
 //print_mat(mat);

 normal_transpose(mat, test_mat);


 //----------//
 hipEventCreate(&start);
 hipEventCreate(&stop);

 hipMemcpy(dev_mat, mat, size, hipMemcpyHostToDevice );

 //-------------//
 hipEventRecord(start);
 transpose<<< (N*N)/N_THREADS, N_THREADS >>>(dev_mat, dev_transp1);
 hipEventRecord(stop);



 hipEventSynchronize(stop);
 float time = 0;
 hipEventElapsedTime(&time, start, stop);

 hipMemcpy(transp1, dev_transp1, size, hipMemcpyDeviceToHost);

 if(test(transp1, test_mat)) printf("correct1\n");
 else printf("not correct1\n");
 int n_threads = N_THREADS; 
 printf("With %d threads time for naive transpose is %fms\n", n_threads, time);

 free(transp1); hipFree(dev_transp1);

 //-------------//

 hipEventRecord(start);
 shared_transpose<<< grid , block >>>(dev_mat, dev_transp2);
 hipEventRecord(stop);


 hipEventSynchronize(stop);
 time = 0;
 hipEventElapsedTime(&time, start, stop);

 hipMemcpy(transp2, dev_transp2, size, hipMemcpyDeviceToHost);

 if(test(transp2, test_mat)) printf("correct2\n");
 else printf("not correct2\n");

 free(transp2); hipFree(dev_transp2);
 //-------------//


 //print_mat(transp2);
 printf("time in shared memory is %fms\n", n_threads, time);

 free(mat); hipFree(dev_mat);

return 0;
}

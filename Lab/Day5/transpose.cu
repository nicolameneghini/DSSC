
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>


#define MAX_ELEM_VALUE 10

#define N 8192
#define BLOCK_X 32
#define BLOCK_Y 32

int n_threads = 1024;


__global__ void transpose(float* mat, float *transp){

 int index = threadIdx.x + blockIdx.x*blockDim.x;
 int x = index%N;
 int y = index/N;
 transp[y*N+x] = mat[x*N+y];

}

__global__ void shared_transpose(float* mat, float *transp){

    __shared__ double temp[BLOCK_X][BLOCK_Y+1];

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    temp[threadIdx.x][threadIdx.y] = mat[index_y * N + index_x];

    __syncthreads();

    transp[index_x * N + index_y] = temp[threadIdx.x][threadIdx.y];

}


void print_mat(float* mat){

 int i, j;

 for(i = 0; i < N; i++){
    for(j = 0; j < N; j++)
     printf("%f ",mat[i*N + j]);

    printf("\n");
  }
}


void randomly_fill_matrix(float *A)
{
  for (unsigned int i = 0; i < N; i++)
  {
    for (unsigned int j = 0; j < N; j++)
    {
      A[i*N + j] = rand() % (2 * MAX_ELEM_VALUE) - MAX_ELEM_VALUE;
    }
  }
}

void normal_transpose(float *mat, float *transp)
{
  for (unsigned int i = 0; i < N; i++)
          for (unsigned int j = 0; j < N; j++)
                  transp[i +j*N] = mat[i*N + j];
}


int test(float *a, float *b)
{

   for (unsigned int i = 0; i < N; i++){
      for (unsigned int j = 0; j < N; j++){
              if(b[i*N + j] != a[i*N + j])
                      return 0;
        }
   }

   return 1;
}
 
int main(void){

 float *mat, *transp1, *transp2, *test_mat;
 float *dev_mat, *dev_transp1, *dev_transp2;
 int size = N*N*sizeof(double);
 hipEvent_t start, stop;
 dim3 grid, block;
 block.x = BLOCK_X;
 block.y = BLOCK_Y;
 grid.x = N/BLOCK_X;
 grid.y = N/BLOCK_Y;

 hipMalloc( (void**)&dev_mat, size );
 hipMalloc( (void**)&dev_transp1, size );
 hipMalloc( (void**)&dev_transp2, size );
 mat = (float*)malloc(size);
 transp1 = (float*)malloc(size);
 transp2 = (float*)malloc(size);
 test_mat = (float*)malloc(size);



 randomly_fill_matrix(mat);
 //print_mat(mat);

 normal_transpose(mat, test_mat);

 //----------//
 hipEventCreate(&start);
 hipEventCreate(&stop);

 hipMemcpy(dev_mat, mat, size, hipMemcpyHostToDevice );

 //-------------//
 hipEventRecord(start);
 transpose<<< (N*N)/n_threads, n_threads  >>>(dev_mat, dev_transp1);
 hipEventRecord(stop);

 hipEventSynchronize(stop);
 float time = 0;
 hipEventElapsedTime(&time, start, stop);

 hipMemcpy(transp1, dev_transp1, size, hipMemcpyDeviceToHost);

 printf("%d %f ", n_threads, time);

 //-------------//

 hipEventRecord(start);
 shared_transpose<<< grid , block >>>(dev_mat, dev_transp2);
 hipEventRecord(stop);


 hipEventSynchronize(stop);
 time = 0;
 hipEventElapsedTime(&time, start, stop);

 hipMemcpy(transp2, dev_transp2, size, hipMemcpyDeviceToHost);

 printf("%f ", n_threads, time);

 if(test(transp1, test_mat)) printf("correct naive ");
 else printf("not correct naive ");

 if(test(transp2, test_mat)) printf("correct shared\n");
 else printf("not correct shared\n");

 //-------------//

 free(transp1); hipFree(dev_transp1);
 free(transp2); hipFree(dev_transp2);
 free(mat); hipFree(dev_mat);

return 0;
}

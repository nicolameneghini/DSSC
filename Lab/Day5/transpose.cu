
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define N 10
#define N_BLOCKS 10
#define N_THREADS 2

__global__ void transpose(int* mat, int *trasp){
 
// int index = threadIdx.x + blockIdx.x*blockDim.x;
 
 int y = blockIdx.x;
 int x = threadIdx.x;
 
 while(x < N){
  trasp[y*N + x] = mat[x*N + y];
  x += blockDim.x;
 }
 
 //int x = index_x%N
 //int y = index_x/N
 //transp[y*N+x] = mat[x*N+y];

}


void print_mat(int* mat){

 int i, j;
 
 for(i = 0; i < N; i++){
    for(j = 0; j < N; j++)
     printf("%d ",mat[i*N + j]);	 
    
    printf("\n");
  }	
}



int main(void){

 int *mat, *transp;
 int *dev_mat, *dev_transp;	
 int size = N*N*sizeof(int);
 int i,j;

 hipMalloc( (void**)&dev_mat, size );
 hipMalloc( (void**)&dev_transp, size );
 
  	 

 mat = (int*)malloc(size);
 transp = (int*)malloc(size);
 
 
 for(i = 0; i < N; i++)
    for(j = 0; j < N; j++)
	    mat[i*N + j] = i;
 	   
 print_mat(mat);
 
 hipMemcpy(dev_mat, mat, size, hipMemcpyHostToDevice );
  
 transpose<<< N, N_THREADS >>>(dev_mat, dev_transp);
 
 hipMemcpy(transp, dev_transp, size, hipMemcpyDeviceToHost);
 
 print_mat(transp);

 free(mat); free(transp);

 hipFree(dev_mat);
 hipFree(dev_transp);


return 0;
}
